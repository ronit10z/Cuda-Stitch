#include "hip/hip_runtime.h"
#include <ctime>
#include <iostream>
#include <chrono>
#include <bitset>

#include "ResponseMapGpuGenerator.cu_incl"


#define CUDA_ERROR_CHECK

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void LaunchKernel(dim3 gridDimensions, dim3 blockDimensions, int* lobeSizesPrecomputed__CUDA, float* gpuIntegralImage, float* determinants, int width ,int height, 
		int numIntervals, int octaveNum, int stepSize, int borderOffset)
{
	GetResponses__CUDA <<<gridDimensions, blockDimensions>>> (lobeSizesPrecomputed__CUDA, gpuIntegralImage, determinants, width, height, numIntervals, octaveNum, stepSize, borderOffset);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
}


__device__ inline float BoxIntegral__CUDA(float* integralImage, int width, int height, int row, int col, int rows, int cols)
{
  int r1 = 	min(row,          height) - 1;
  int c1 = 	min(col,          width)  - 1;
  int r2 = 	min(row + rows,   height) - 1;
  int c2 = 	min(col + cols,   width)  - 1;

  float A(0.0f), B(0.0f), C(0.0f), D(0.0f);
  // width step is in mempry how much each row is(number of bytes in a row, like actually)
  // pretty much the same as in the sequential version, now just need to interface directly with the data struct :(
  if (r1 >= 0 && c1 >= 0) A = integralImage[r1 * width + c1];
  if (r1 >= 0 && c2 >= 0) B = integralImage[r1 * width + c2];
  if (r2 >= 0 && c1 >= 0) C = integralImage[r2 * width + c1];
  if (r2 >= 0 && c2 >= 0) D = integralImage[r2 * width + c2];

  return max(0.f, A - B - C + D);
}

__global__ void GetResponses__CUDA(int* lobeSizesPrecomputed__CUDA, float* integralImage, float* determinants, int width ,int height, 
		int numIntervals, int octaveNum, int stepSize, int borderOffset)

{
	int integralImageRow = (blockIdx.y * blockDim.y + threadIdx.y) * stepSize;

	int blocksPerInterval_x = gridDim.x / numIntervals;
	int integralImageCol = (blockIdx.x % blocksPerInterval_x);
	integralImageCol *= blockDim.x; // getting down to thread idx
	integralImageCol += threadIdx.x;
	integralImageCol *= stepSize;

  if (integralImageCol >= width || integralImageRow >= height) 
  {
    return;
  }

	int lobeSizesPrecomputedOffset = blockIdx.x / blocksPerInterval_x + (octaveNum > 0) * 2;

	const int lobeSize = lobeSizesPrecomputed__CUDA[octaveNum * numIntervals + lobeSizesPrecomputedOffset];;
	const int filterSize = lobeSize * 3;
	const int borderSize = filterSize / 2 + 1;

	float Dxx = BoxIntegral__CUDA(integralImage, width, height, integralImageRow - lobeSize + 1, integralImageCol - borderSize, 2*lobeSize - 1, filterSize)
          - BoxIntegral__CUDA(integralImage, width, height, integralImageRow - lobeSize + 1, integralImageCol - lobeSize / 2, 2*lobeSize - 1, lobeSize)*3;
  float Dyy = BoxIntegral__CUDA(integralImage, width, height, integralImageRow - borderSize, integralImageCol - lobeSize + 1, filterSize, 2*lobeSize - 1)
          - BoxIntegral__CUDA(integralImage, width, height, integralImageRow - lobeSize / 2, integralImageCol - lobeSize + 1, lobeSize, 2*lobeSize - 1)*3;
  float Dxy = + BoxIntegral__CUDA(integralImage, width, height, integralImageRow - lobeSize, integralImageCol + 1, lobeSize, lobeSize)
            + BoxIntegral__CUDA(integralImage, width, height, integralImageRow + 1, integralImageCol - lobeSize, lobeSize, lobeSize)
            - BoxIntegral__CUDA(integralImage, width, height, integralImageRow - lobeSize, integralImageCol - lobeSize, lobeSize, lobeSize)
            - BoxIntegral__CUDA(integralImage, width, height, integralImageRow + 1, integralImageCol + 1, lobeSize, lobeSize);

  float inverseArea = 1.f / (filterSize * filterSize);
  Dxx *= inverseArea;
  Dyy *= inverseArea;
  Dxy *= inverseArea;

  float determinant = (Dxx * Dyy - 0.81f * Dxy * Dxy);

  unsigned int computed_interval = octaveNum * numIntervals + lobeSizesPrecomputedOffset;
  unsigned int interval_start_index = computed_interval * width * height;
  unsigned int determinant_pixel_index = interval_start_index + (integralImageRow * width + integralImageCol);

  // if(octaveNum == 1 && computed_interval == 4) printf("%d %d %d %d\n", determinant_pixel_index - 3 *(width * height), integralImageRow, integralImageCol, computed_interval);
  determinants[determinant_pixel_index] = determinant;
}	